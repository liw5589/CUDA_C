// %%cu Google Colap 에서 실행할 때 


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define SIZE_OF_ARR 3

void  CUDA_ERR_CHECK(hipError_t err, int lineNo)
{
    if (err != hipSuccess)
    {
        printf("Cuda Err : %s , Check Line No : %d\n", hipGetErrorString(err), lineNo);
    }
}

__global__ void MultipleFunctionInGPU(int *DEVICE_arrA, int *DEVICE_arrB, int *DEVICE_arrC)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < SIZE_OF_ARR && j < SIZE_OF_ARR)
    {
        // DEVICE_arrC[i] = DEVICE_arrA[i * SIZE_OF_ARR + j] + DEVICE_arrB[j];
        atomicAdd(&DEVICE_arrC[i], DEVICE_arrA[i * SIZE_OF_ARR + j] * DEVICE_arrB[j]);
    }
}

int main()
{
    int *HOST_arrA, *HOST_arrB, *HOST_arrC;       // CPU에서 사용할 메모리
    int *DEVICE_arrA, *DEVICE_arrB, *DEVICE_arrC; // GPU에서 사용할 메모리

    hipError_t err;

    HOST_arrA = (int *)malloc(SIZE_OF_ARR * SIZE_OF_ARR * sizeof(int)); // 1000 * 1000
    HOST_arrB = (int *)malloc(SIZE_OF_ARR * sizeof(int));               // 1000 * 1
    HOST_arrC = (int *)malloc(SIZE_OF_ARR * sizeof(int));               // 1000 * 1

    // 변수 초기화
    for (int i = 0; i < SIZE_OF_ARR; i++)
    {
        HOST_arrB[i] = i + 1;
        for (int j = 0; j < SIZE_OF_ARR; j++)
        {
            HOST_arrA[i * SIZE_OF_ARR + j] = 1;
        }
    }

    // GPU Memory에 할당하는 변수
    err = hipMalloc(&DEVICE_arrA, SIZE_OF_ARR * SIZE_OF_ARR * sizeof(int));
    CUDA_ERR_CHECK(err, __LINE__);
    err = hipMemcpy(DEVICE_arrA, HOST_arrA, SIZE_OF_ARR * SIZE_OF_ARR * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK(err, __LINE__);

    err = hipMalloc(&DEVICE_arrB, SIZE_OF_ARR * sizeof(int));
    CUDA_ERR_CHECK(err, __LINE__);
    err = hipMemcpy(DEVICE_arrB, HOST_arrB, SIZE_OF_ARR * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK(err, __LINE__);

    err = hipMalloc(&DEVICE_arrC, SIZE_OF_ARR * sizeof(int));
    CUDA_ERR_CHECK(err, __LINE__);
    err = hipMemcpy(DEVICE_arrC, HOST_arrC, SIZE_OF_ARR * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK(err, __LINE__);

    dim3 DimGrid(3, 3);
    dim3 DimBlock(SIZE_OF_ARR / 3, SIZE_OF_ARR / 3);

    MultipleFunctionInGPU<<<DimGrid, DimBlock>>>(DEVICE_arrA, DEVICE_arrB, DEVICE_arrC);
    hipDeviceSynchronize();

    hipMemcpy(HOST_arrC, DEVICE_arrC, SIZE_OF_ARR * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < SIZE_OF_ARR; i++)
    {
        printf("Host C [%d] : %d \n", i, HOST_arrC[i]);
    }

    free(HOST_arrA);
    hipFree(DEVICE_arrA);
    free(HOST_arrB);
    hipFree(DEVICE_arrB);
    free(HOST_arrC);
    hipFree(DEVICE_arrC);

    hipDeviceReset();
}